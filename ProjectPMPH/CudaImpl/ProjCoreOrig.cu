#include "hip/hip_runtime.h"
#include "ProjHelperFun.h"
#include "Constants.h"
#include "TridagPar.h"
#include "kernels.cu"
#include "ProjHelperFun.cu"
#include "cudaErrHandling.cu"
#include <vector>

// void printArray(vector<REAL> arr) {
//     printf("[");
//     for (const auto& elem : arr) {
//         printf("%f, ", elem);
//     }
//     printf("]\n");
// }

void
rollback( const unsigned g, PrivGlobs& globs, vector<vector<vector<REAL > > >& myResult, REAL* d_myResult, const unsigned int outer) {
    unsigned numX = globs.myX.size(),
             numY = globs.myY.size();
    unsigned numZ = max(numX,numY);
    unsigned x, y, i, j;
    REAL dtInv = 1.0/(globs.myTimeline[g+1]-globs.myTimeline[g]);

    vector<vector<vector<REAL> > > u(outer, vector<vector<REAL> > (numY, vector<REAL>(numX))); // [outer][numY][numX]
    vector<vector<vector<REAL> > > v(outer, vector<vector<REAL> > (numX, vector<REAL>(numY))); // [outer][numX][numY]
    vector<REAL> a(numZ); // [max(numX,numY)]
    vector<REAL> b(numZ); // [max(numX,numY)]
    vector<REAL> c(numZ); // [max(numX,numY)]
    vector<vector<REAL> > _y(outer, vector<REAL>(numZ)); // [outer][max(numX,numY)]

    vector<vector<REAL> > yy(outer, vector<REAL>(numZ));  // temporary used in tridag  // [max(numX,numY)]

    REAL* d_myVarX;
    REAL* d_myVarY;
    REAL* d_myDxx;
    REAL* d_myDyy;
    REAL* d_u;
    REAL* d_v;


    /*      Allocate device memory  */
    unsigned long myVarSize = numX * numY * sizeof(REAL);
    hipMalloc((void**)&d_myVarX, myVarSize);
    hipMalloc((void**)&d_myVarY, myVarSize);
    unsigned long myDxxSize = numX * 4 * sizeof(REAL);
    unsigned long myDyySize = numY * 4 * sizeof(REAL);
    hipMalloc((void**)&d_myDxx, myDxxSize);
    hipMalloc((void**)&d_myDyy, myDyySize);
    unsigned long myResultSize = outer * numX * numY * sizeof(REAL);
    hipMalloc((void**)&d_u, myResultSize);
    hipMalloc((void**)&d_v, myResultSize);

    /*      Copy data to device  */
    copy2DVec(d_myVarX, globs.myVarX, hipMemcpyHostToDevice);
    copy2DVec(d_myVarY, globs.myVarY, hipMemcpyHostToDevice);
    copy2DVec(d_myDxx, globs.myDxx, hipMemcpyHostToDevice);
    copy2DVec(d_myDyy, globs.myDyy, hipMemcpyHostToDevice);
    copy3DVec(d_myResult, myResult, hipMemcpyHostToDevice);

    /*      Call kernel  */
    unsigned dim = 32;
    int dimO = ceil( ((float)outer) / dim );
    int dimX = ceil( ((float)numX) / dim );

    dim3 block(dim, dim, 1), grid(dimO, dimX, 1);

    initUAndV2Dim<<<grid, block>>>(d_u, d_v, d_myVarX, d_myVarY, d_myDxx, d_myDyy, d_myResult, outer, numX, numY, dtInv);

    /*      Copy data back to host */
    copy3DVec(d_u, u, hipMemcpyDeviceToHost);
    copy3DVec(d_v, v, hipMemcpyDeviceToHost);


    /*      Free device memory  */
    hipFree(d_myVarX);
    hipFree(d_myVarY);
    hipFree(d_myDxx);
    hipFree(d_myDyy);
    hipFree(d_u);
    hipFree(d_v);

    for( unsigned o = 0; o < outer; ++ o ) {
        //  implicit x
        for(y = 0; y < numY; y++) {
            for(x = 0; x < numX; x++) {  // here a, b,c should have size [numX]
                a[x] =       - 0.5*(0.5*globs.myVarX[x][y]*globs.myDxx[x][0]);
                b[x] = dtInv - 0.5*(0.5*globs.myVarX[x][y]*globs.myDxx[x][1]);
                c[x] =       - 0.5*(0.5*globs.myVarX[x][y]*globs.myDxx[x][2]);
            }
            // here yy should have size [numX]
            tridagPar(a,b,c,u[o][y],numX,u[o][y],yy[o]);
        }

        //  implicit y
        for(x = 0; x < numX; x++) {
            for(y = 0; y < numY; y++) {  // here a, b, c should have size [numY]
                a[y] =       - 0.5*(0.5*globs.myVarY[x][y]*globs.myDyy[y][0]);
                b[y] = dtInv - 0.5*(0.5*globs.myVarY[x][y]*globs.myDyy[y][1]);
                c[y] =       - 0.5*(0.5*globs.myVarY[x][y]*globs.myDyy[y][2]);
            }

            for(y = 0; y < numY; y++) {
                _y[o][y] = dtInv*u[o][y][x] - 0.5*v[o][x][y];
            }

            // here yy should have size [numY]
            tridagPar(a,b,c,_y[o],numY,myResult[o][x],yy[o]);
        }
    }
}

void   run_OrigCPU(
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t,
                const REAL&           alpha,
                const REAL&           nu,
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {

    PrivGlobs    globs(numX, numY, numT);
    initGrid(s0,alpha,nu,t, numX, numY, numT, globs);
    initOperator(globs.myX,globs.myDxx);
    initOperator(globs.myY,globs.myDyy);

    vector<vector<vector<REAL> > > myResult(outer, vector<vector<REAL > >(numX, vector<REAL> (numY)));

    REAL *d_myResult;
    // Compute myResult from a 2d kernel
    {
        cudaErrchkAPI(hipMalloc((void**)&d_myResult, outer * numX * numY * sizeof(REAL)));

        REAL *d_myX;
        cudaErrchkAPI(hipMalloc((void**)&d_myX, numX * sizeof(REAL)));
        cudaErrchkAPI(hipMemcpy(d_myX, globs.myX.data(), numX * sizeof(REAL), hipMemcpyHostToDevice));

        int T =32;
        int dimy = ceil(((float)outer) / T);
        int dimx = ceil(((float)numX) / T);
        dim3 block(T, T, 1), grid(dimx, dimy, 1);

        myResultKernel2D<<<grid, block>>>(outer, numX, numY, d_myX, d_myResult);
        cudaErrchkKernelAndSync();

        copy3DVec(d_myResult, myResult, hipMemcpyDeviceToHost);

        cudaErrchkAPI(hipFree(d_myX));
    }

    for(int g = globs.myTimeline.size()-2;g>=0;--g) {
        for(unsigned x = 0; x < globs.myX.size(); ++x) {
            for(unsigned y = 0; y < globs.myY.size(); ++y) {
                globs.myVarX[x][y] = exp(2.0*(  beta*log(globs.myX[x])
                                              + globs.myY[y]
                                              - 0.5*nu*nu*globs.myTimeline[g] )
                                        );
                globs.myVarY[x][y] = exp(2.0*(  alpha*log(globs.myX[x])
                                              + globs.myY[y]
                                              - 0.5*nu*nu*globs.myTimeline[g] )
                                        ); // nu*nu
            }
        }

        rollback(g, globs, myResult, d_myResult, outer);
        // g = -1;
    }
    hipFree(d_myResult);
    for( unsigned o = 0; o < outer; ++o ) {
        res[o] = myResult[o][globs.myXindex][globs.myYindex];
    }
}

//#endif // PROJ_CORE_ORIG

#include "hip/hip_runtime.h"
__global__ void initUAndV2Dim (
                    REAL* u,
                    REAL* v,
                    REAL* myVarX,
                    REAL* myVarY,
                    REAL* myDxx,
                    REAL* myDyy,
                    REAL* myResult,
                    unsigned outer,
                    unsigned numX,
                    unsigned numY,
                    REAL dtInv
) {
    int o = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;

    if (o < outer && x < numX) {
        for(unsigned y = 0; y < numY; y++) {

            // explicit x
            unsigned uIdx_oyx = (o*numY * numX) + (y*numX) + x;

            u[uIdx_oyx] = dtInv * myResult[(o*numX*numY) + (x*numY) + y];

            if (x > 0) {
                u[uIdx_oyx] += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 0])
                            * myResult[(o*numX*numY) + ((x-1)*numY) + y];
            }
            u[uIdx_oyx] += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 1])
                        * myResult[(o*numX*numY) + (x*numY) + y];
            if (x < numX - 1) {
                u[uIdx_oyx] += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 2])
                            * myResult[(o*numX*numY) + ((x+1)*numY) + y];
            }

            // explicit y
            unsigned vIdx_oxy = (o*numX*numY) + (x*numY) + y;
            v[vIdx_oxy] = 0.0;
            if(y > 0) {
                v[vIdx_oxy] += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 0])
                    *  myResult[(o*numX*numY) + (x*numY) + y-1];
            }
            v[vIdx_oxy] += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 1])
                *  myResult[(o*numX*numY) + (x*numY) + y];
            if(y < numY - 1) {
                v[vIdx_oxy] += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 2])
                    *  myResult[(o*numX*numY) + (x*numY) + y+1];
            }
            u[uIdx_oyx] += v[vIdx_oxy];
        }
    }
}

__global__ void myResultKernel2D(unsigned int outer, unsigned int numX, unsigned int numY, REAL * myX, REAL * myResult) {
	int o = threadIdx.x + blockDim.x*blockIdx.x;
  	int x = threadIdx.y + blockDim.y*blockIdx.y;

  	if (o < outer && x < numX) {
        for(unsigned y = 0; y < numY; y++) {
            myResult[o * numX * numY + x * numY + y] = max(myX[x]-(0.001*o), (REAL)0.0);;
        }
	}
}

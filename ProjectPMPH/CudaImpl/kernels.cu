#include "hip/hip_runtime.h"
__global__ void initUAndV2Dim (
                    REAL* u,
                    REAL* v,
                    REAL* myVarX,
                    REAL* myVarY,
                    REAL* myDxx,
                    REAL* myDyy,
                    REAL* myResult,
                    unsigned outer,
                    unsigned numX,
                    unsigned numY,
                    REAL dtInv
) {
    int o = blockIdx.x*blockDim.x + threadIdx.x;
    int x = blockIdx.y*blockDim.y + threadIdx.y;

    if (o < outer && x < numX) {
        for(unsigned y = 0; y < numY; y++) {

            // explicit x
            REAL u_new = dtInv * myResult[(o*numX*numY) + (x*numY) + y];

            if (x > 0) {
                u_new += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 0])
                            * myResult[(o*numX*numY) + ((x-1)*numY) + y];
            }
            u_new += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 1])
                        * myResult[(o*numX*numY) + (x*numY) + y];
            if (x < numX - 1) {
                u_new += 0.5*( 0.5*myVarX[(x*numY) + y] * myDxx[(x*4) + 2])
                            * myResult[(o*numX*numY) + ((x+1)*numY) + y];
            }

            // explicit y
            REAL v_new = 0.0;
            if(y > 0) {
                v_new += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 0])
                    *  myResult[(o*numX*numY) + (x*numY) + y-1];
            }
            v_new += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 1])
                *  myResult[(o*numX*numY) + (x*numY) + y];
            if(y < numY - 1) {
                v_new += ( 0.5*myVarY[(x*numY) + y] * myDyy[(y*4) + 2])
                    *  myResult[(o*numX*numY) + (x*numY) + y+1];
            }
            v[(x*numY*outer) + (y*outer) + o] = v_new;
            u[(y*numX*outer) + (x*outer) + o] = u_new + v_new;
        }
    }
}

__global__ void myResultKernel2D(unsigned int outer, unsigned int numX, unsigned int numY, REAL *myX, REAL *myResult) {
	int o = threadIdx.x + blockDim.x*blockIdx.x;
  	int x = threadIdx.y + blockDim.y*blockIdx.y;

  	if (o < outer && x < numX) {
  		REAL v = max(myX[x]-(0.001*o), (REAL)0.0);
        for(unsigned y = 0; y < numY; y++) {
            myResult[o * numX * numY + x * numY + y] = v;
        }
	}
}

__global__ void myVarXYKernel(
	unsigned int numX, unsigned int numY,
	REAL beta, REAL nu2t, REAL alpha,
	REAL *myX, REAL *myY,
	REAL *myVarX, REAL *myVarY
	) {
	int x = threadIdx.x + blockDim.x*blockIdx.x;
  	int y = threadIdx.y + blockDim.y*blockIdx.y;

  	if (x < numX && y < numY) {
        myVarX[x * numY + y] = exp(2.0*(  beta*log(myX[x])
	                                      + myY[y]
	                                      - nu2t )
	                                );
        myVarY[x * numY + y] = exp(2.0*(  alpha*log(myX[x])
	                                      + myY[y]
	                                      - nu2t )
	                                ); // nu*nu
	}
}

__global__ void buildResultKernel(
	unsigned int outer, unsigned int numX, unsigned int numY,
	unsigned int myXindex, unsigned int myYindex,
	REAL *res, REAL *myResult
	) {
	const unsigned int o = threadIdx.x + blockDim.x * blockIdx.x;

	if (o < outer) {
        res[o] = myResult[o * numX * numY + myXindex * numY + myYindex];
    }
}

__device__ inline void tridagDevice1(
    REAL*   a,   // size [n]
    REAL*   b,   // size [n]
    REAL*   c,   // size [n]
    REAL*   r,   // size [n] u
    const int n,
    REAL*   u,   // size [n] u
    REAL*   yy,   // size [n] temporary
    const int mult,  // multiplier to index into arrays
    const int multU
) {
    int    i;
    REAL   beta;

    u[0]  = r[0];
    yy[0] = b[0];

    for(i=1; i<n; i++) {
        beta  = a[i * mult] / yy[(i-1) * mult];

        yy[i * mult] = b[i * mult] - beta*c[(i-1) * mult];
        u[i * multU]  = r[i*multU] - beta*u[(i-1) * multU];
    }

    // X) this is a backward recurrence
    u[(n-1)*multU] = u[(n-1)*multU] / yy[(n-1) * mult];
    for(i=n-2; i>=0; i--) {
        u[i*multU] = (u[i*multU] - c[i * mult]*u[(i+1)*multU]) / yy[i * mult];
    }
}

__global__ void tridag1(
    unsigned int outer, unsigned int numX, unsigned int numY, unsigned int numZ,
    REAL *a, REAL *b, REAL *c,
    REAL dtInv,
    REAL *myVarX, REAL *myDxx,
    REAL *u, REAL *yy
    ) {
    int o = threadIdx.x + blockDim.x*blockIdx.x;
    int y = threadIdx.y + blockDim.y*blockIdx.y;

    if (o < outer && y < numY) {
        for(unsigned x = 0; x < numX; x++) {
            // here a, b,c should have size [numX]
            a[o + y * outer + x * numZ * outer] =       - 0.5*(0.5*myVarX[x * numY + y]*myDxx[x * 4 + 0]);
            b[o + y * outer + x * numZ * outer] = dtInv - 0.5*(0.5*myVarX[x * numY + y]*myDxx[x * 4 + 1]);
            c[o + y * outer + x * numZ * outer] =       - 0.5*(0.5*myVarX[x * numY + y]*myDxx[x * 4 + 2]);
        }

        // here yy should have size [numX]
        tridagDevice1(
            a + (o + y * outer),
            b + (o + y * outer),
            c + (o + y * outer),
            u + (o + y * numX * outer),
            numX,
            u + (o + y * numX * outer),
            yy+ (o + y * outer),
            numZ * outer,
            outer
        );
    }
}

__device__ inline void tridagDevice2(
    REAL*   a,   // size [n]
    REAL*   b,   // size [n]
    REAL*   c,   // size [n]
    REAL*   r,   // size [n] _y
    const int n,
    REAL*   u,   // size [n] myResult
    REAL*   yy,   // size [n] temporary
    const int mult // multiplier to index into arrays
) {
    int    i;
    REAL   beta;

    u[0]  = r[0];
    yy[0] = b[0];

    for(i=1; i<n; i++) {
        beta  = a[i * mult] / yy[(i-1) * mult];

        yy[i * mult] = b[i * mult] - beta*c[(i-1) * mult];
        u[i]  = r[i] - beta*u[(i-1)];
    }

    // X) this is a backward recurrence
    u[(n-1)] = u[(n-1)] / yy[(n-1) * mult];
    for(i=n-2; i>=0; i--) {
        u[i] = (u[i] - c[i * mult]*u[(i+1)]) / yy[i * mult];
    }
}

__global__ void tridag2(
    unsigned int outer, unsigned int numX, unsigned int numY, unsigned int numZ,
    REAL *a, REAL *b, REAL *c,
    REAL dtInv,
    REAL *myVarY, REAL *myDyy,
    REAL *u, REAL *v, REAL *yy, REAL *_y, REAL *myResult
    ) {

    int o = threadIdx.x + blockDim.x*blockIdx.x;
    int x = threadIdx.y + blockDim.y*blockIdx.y;

    if (o < outer && x < numX) {
        int ox_idx_zo = o + x * numZ * outer;
        for(unsigned y = 0; y < numY; y++) {
            // here a, b, c should have size [numY]
            a[ox_idx_zo + y * outer] =       - 0.5*(0.5*myVarY[x * numY + y]*myDyy[y * 4 + 0]);
            b[ox_idx_zo + y * outer] = dtInv - 0.5*(0.5*myVarY[x * numY + y]*myDyy[y * 4 + 1]);
            c[ox_idx_zo + y * outer] =       - 0.5*(0.5*myVarY[x * numY + y]*myDyy[y * 4 + 2]);
        }

        for(unsigned y = 0; y < numY; y++) {
            _y[o * numZ * numZ + x * numZ + y] = dtInv*u[(y*numX*outer) + (x*outer) + o] - 0.5*v[x * numY * outer + y * outer + o];
        }

        // here yy should have size [numY]
        tridagDevice2(
            a + (ox_idx_zo),
            b + (ox_idx_zo),
            c + (ox_idx_zo),
            _y + (o * numZ * numZ + x * numZ),
            numY,
            myResult + (o * numX * numY + x * numY),
            yy + (ox_idx_zo),
            outer
        );
    }
}
